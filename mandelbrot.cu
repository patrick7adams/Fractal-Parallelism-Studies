#include "hip/hip_runtime.h"
#ifndef MANDELBROT_CPP
#define MANDELBROT_CPP
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

const int resX = 1000;
const int resY = 1000;
const int iterations = 1000;
const int totalPoints = resX * resY;
// Things to do with this:
// CUDA Multithreading
// Heterogeneous Processing
/**
 * @brief A representation of a point on the complex plane.
 */
struct Point {
    double r; // Real component
    double i; // Imaginary component
};

struct Bounds {
    Point tl;
    Point br;
    double lenX;
    double lenY;
};

/**
 * @brief Checks if a point is a part of the mandelbrot set.
 *
 * @param Point p, the point at which to check if it is a member of the mandelbrot set.
 * @param int numIterations, the maximum number of iterations of the mandelbrot equation to iterate through
 * before terminating.
 * @return int, the number of iterations that the point underwent before either the maximum number of
 * iterations is reached or the point diverges.
 */
__global__ void checkMandelbrot(int *iter, const Point& p) {
    Point lastP = { 0, 0 };
    while (*iter < iterations && lastP.r * lastP.r + lastP.i * lastP.i <= 4) {
        // squares the real and the imaginary components and adds them with the first iteration.
        double tempr = lastP.r;
        lastP.r = (lastP.r * lastP.r - lastP.i * lastP.i) + p.r;
        lastP.i = 2 * tempr * lastP.i + p.i;
        *iter++;
    }
}

__global__ void initializePointsAndVertices(float* vertices, Point *points, Bounds *bounds) {
    for (int i = 0; i < resX; i++) {
        for (int k = 0; k < resY; k++) {
            Point p = Point{ *bounds.tl.r - (double)i / resX * lenX, *tl.i - (double)k / resY * lenY };

            vertices[(i * resX + k) * 2] = ((float)(i * 2.0 - resX)) / (resX);
            vertices[(i * resX + k) * 2 + 1] = ((float)(k * 2.0 - resY)) / (resY);
        }
    }
}

/**
 * @brief Generates a file of iterations from the mandelbrot set.
 *
 * @param Point topLeft, the top left point of the screen to render.
 * @param Point bottomRight, the bottom right point of the screen to render.
 * @param int numIterations, the maximum number of iterations of the mandelbrot equation to iterate through before terminating.
 * @param int resolutionX, the length of the screen in the horizontal plane.
 * @param int resolutionY, the length of the screen in the vertical plane.
 */
void genMandelbrot(float* vertices, float* colors,  Bounds& bounds) {
    printf("Generating set...\n");
    
    int* iter = new int[totalPoints];
    Point* points = new Point[totalPoints];

    initializePointsAndVertices(vertices, points, bounds);
            int* pIter = 0;

            checkMandelbrot(pIter, p);

            iter[i*resX+k] = pIter;
            // Gets normalized vertices for rendering.
            
    // Coloring in the set.
    printf("Coloring set...\n");
    for (int i = 0; i < totalPoints; i++) {
        if (iter[i] == iterations) {
            colors[i * 3] = 0.0;
            colors[i * 3 + 1] = 0.0;
            colors[i * 3 + 2] = 0.0;
        }
        else {
            float val = fmod(pow(((float)iter[i] / iterations) * 50, 1.5), 1);
            colors[i * 3] = val;
            colors[i * 3 + 1] = val;
            colors[i * 3 + 2] = 1.0 - val;
        }
    }
    delete[] iter;
}
#endif