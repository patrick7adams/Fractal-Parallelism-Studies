#include "hip/hip_runtime.h"
#include "mandel_cuda.cuh"

__global__ void cuda_check_points(int iLen, int* iter, struct Point* p) {
    int iStart = blockIdx.x * blockDim.x + threadIdx.x;
    double tempr;
    int it;
    if(iStart*iLen < totalPoints) {
        for (int i = iStart*iLen; i < iStart*iLen + iLen; i++) {
            struct Point lastP = { 0.0, 0.0 };
            it = 0;
            while (it < maxIterations && lastP.r * lastP.r + lastP.i * lastP.i <= 4) {
                // squares the real and the imaginary components and adds them with the first iteration.
                tempr = lastP.r;
                lastP.r = (lastP.r * lastP.r - lastP.i * lastP.i) + p[i].r;
                lastP.i = 2.0 * tempr * lastP.i + p[i].i;
                it++;
            }
            iter[i] = it;
            // printf("Iteration %d at Point (%f, %f): %d\n", i, p[i].r, p[i].i, iter[i]);
        }
    }
}

double cuda_gen(int* iter, const struct Point* p) {
    int* cuda_iter;
    hipMalloc(&cuda_iter, sizeof(int) * totalPoints);
    struct Point* cuda_p;
    hipMalloc(&cuda_p, sizeof(struct Point)*totalPoints);
    hipMemcpy(cuda_p, p, sizeof(struct Point)*totalPoints, hipMemcpyHostToDevice);

    hipEvent_t start,end;
    float timeElapsed;

    int numThreads = 128;
    int numBlocks = 256;
    int iLen = totalPoints / (numThreads*numBlocks)+1;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    cuda_check_points<<<numThreads, numBlocks>>>(iLen, cuda_iter, cuda_p);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    hipMemcpy(iter, cuda_iter, sizeof(int)*totalPoints, hipMemcpyDeviceToHost);
    hipFree(cuda_iter);
    hipFree(cuda_p);

    hipEventElapsedTime(&timeElapsed, start, end);
    return ((double)timeElapsed)/1000.0; // convert ms to s
}